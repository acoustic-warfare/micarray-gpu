#include "hip/hip_runtime.h"


#include <stdio.h>

extern "C" void cudaBeamWrapper(int *res, const int *first, const int *last, int n_bytes);

__global__ void beamKernel(int *res, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        res[i] = a[i] * b[i];
    }
}

// Cuda Wrapper for `beamKernel` used by C or Cython code
void cudaBeamWrapper(int *res, const int *first, const int *last, int n_bytes)
{
    // Setup buffers for GPU
    int *dev_res = nullptr;
    int *dev_first = nullptr;
    int *dev_last = nullptr;

    // Allocate memory on GPU for three vectors
    hipMalloc((void **)&dev_res, n_bytes * sizeof(int));
    hipMalloc((void **)&dev_first, n_bytes * sizeof(int));
    hipMalloc((void **)&dev_last, n_bytes * sizeof(int));

    // Copy allocated host memory to device
    hipMemcpy(dev_first, first, n_bytes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_last, last, n_bytes * sizeof(int), hipMemcpyHostToDevice);

    // Compute the result using one thread per element in vector
    // 2 is number of computational blocks and (n_bytes + 1) / 2 is a number of threads in a block
    beamKernel<<<2, (n_bytes + 1) / 2>>>(dev_res, dev_first, dev_last, n_bytes);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(res, dev_res, n_bytes * sizeof(int), hipMemcpyDeviceToHost);

    // Release allocated memory
    hipFree(dev_res);
    hipFree(dev_first);
    hipFree(dev_last);

    hipDeviceReset();
}



// int main(int argc, char **argv)
// {
//     const int arraySize = BYTES;
//     int res[arraySize] = {0};
//     int first[arraySize];
//     int last[arraySize];

//     // Inititate random values
//     int i;
//     for (i = 0; i < BYTES; i++)
//     {
//         first[i] = rand();
//     }
//     for (i = 0; i < BYTES; i++)
//     {
//         last[i] = rand();
//     }
    
//     cudaBeamWrapper(res, first, last, arraySize);
//     int loop;
//     for (loop = 0; loop < BYTES; loop++)
//         printf("%d ", res[loop]);
//     printf("\n");
//     cudaDeviceReset();

//     return 0;
// }